// CUDA optimized kernels for FastCall2 GPU acceleration
// Optimized for NVIDIA GPUs using CUDA-specific features

#include <hip/hip_runtime.h>


// CUDA optimized allele counting kernel
__global__ void countAllelesCUDA(const char* pileupData,
                                const int* positions,
                                const char* refBases,
                                int* alleleCounts,
                                const int dataLength,
                                const int numPositions,
                                const int maxAlleles) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int lid = threadIdx.x;

    // Use shared memory for counting optimization (CUDA-specific)
    __shared__ int sharedCounts[256 * 6]; // 6 base types: A,C,G,T,-,+

    if (gid >= numPositions) return;

    // Initialize shared memory counters
    for (int i = lid; i < 256 * 6; i += blockDim.x) {
        sharedCounts[i] = 0;
    }
    __syncthreads();

    // Process pileup data
    int position = positions[gid];
    int startIdx = gid * dataLength;

    // Count various alleles
    for (int i = 0; i < dataLength; i++) {
        char base = pileupData[startIdx + i];
        int alleleType = -1;

        switch(base) {
            case 'A': case 'a': alleleType = 0; break;
            case 'C': case 'c': alleleType = 1; break;
            case 'G': case 'g': alleleType = 2; break;
            case 'T': case 't': alleleType = 3; break;
            case '-': alleleType = 4; break;
            case '+': alleleType = 5; break;
        }

        if (alleleType >= 0) {
            atomicAdd(&sharedCounts[lid * 6 + alleleType], 1);
        }
    }

    __syncthreads();

    // Merge shared counts to global memory
    if (lid == 0) {
        for (int i = 0; i < 6; i++) {
            int totalCount = 0;
            for (int j = 0; j < blockDim.x; j++) {
                totalCount += sharedCounts[j * 6 + i];
            }
            alleleCounts[gid * 6 + i] = totalCount;
        }
    }
}

// CUDA optimized genotype calculation kernel
__global__ void calculateGenotypesCUDA(const int* alleleCounts,
                                      const char* refBases,
                                      float* genotypeProbabilities,
                                      int* bestGenotypes,
                                      const int numPositions,
                                      const float qualityThreshold) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numPositions) return;

    int baseIdx = gid * 6;

    // Get allele counts
    int countA = alleleCounts[baseIdx + 0];
    int countC = alleleCounts[baseIdx + 1];
    int countG = alleleCounts[baseIdx + 2];
    int countT = alleleCounts[baseIdx + 3];
    int countDel = alleleCounts[baseIdx + 4];
    int countIns = alleleCounts[baseIdx + 5];

    int totalDepth = countA + countC + countG + countT + countDel + countIns;

    if (totalDepth == 0) {
        bestGenotypes[gid] = -1; // Invalid genotype
        return;
    }

    // Calculate frequencies using CUDA math functions
    float freqA = __fdividef((float)countA, (float)totalDepth);
    float freqC = __fdividef((float)countC, (float)totalDepth);
    float freqG = __fdividef((float)countG, (float)totalDepth);
    float freqT = __fdividef((float)countT, (float)totalDepth);

    // Find the two highest frequency alleles
    float maxFreq1 = 0.0f, maxFreq2 = 0.0f;
    int allele1 = -1, allele2 = -1;

    float freqs[4] = {freqA, freqC, freqG, freqT};

    for (int i = 0; i < 4; i++) {
        if (freqs[i] > maxFreq1) {
            maxFreq2 = maxFreq1;
            allele2 = allele1;
            maxFreq1 = freqs[i];
            allele1 = i;
        } else if (freqs[i] > maxFreq2) {
            maxFreq2 = freqs[i];
            allele2 = i;
        }
    }

    // Calculate genotype probabilities (simplified likelihood calculation)
    float homozygousProb = maxFreq1 * maxFreq1;
    float heterozygousProb = 2.0f * maxFreq1 * maxFreq2;

    // Select the most likely genotype
    if (homozygousProb > heterozygousProb) {
        bestGenotypes[gid] = allele1 * 10 + allele1; // Homozygous (AA, CC, GG, TT)
        genotypeProbabilities[gid] = homozygousProb;
    } else {
        bestGenotypes[gid] = allele1 * 10 + allele2; // Heterozygous (AC, AG, AT, etc.)
        genotypeProbabilities[gid] = heterozygousProb;
    }
}

// CUDA optimized probability calculation kernel using Bayesian method
__global__ void calculateProbabilitiesCUDA(const int* alleleCounts,
                                          const float* qualityScores,
                                          float* posteriorProbabilities,
                                          const int numPositions,
                                          const float priorProb) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numPositions) return;

    int baseIdx = gid * 6;

    // Get allele counts
    int counts[6];
    for (int i = 0; i < 6; i++) {
        counts[i] = alleleCounts[baseIdx + i];
    }

    int totalDepth = counts[0] + counts[1] + counts[2] + counts[3] + counts[4] + counts[5];

    if (totalDepth == 0) {
        posteriorProbabilities[gid] = 0.0f;
        return;
    }

    // Calculate likelihood function (binomial distribution approximation)
    float likelihood = 1.0f;
    float qualityScore = qualityScores ? qualityScores[gid] : 30.0f; // Default quality score
    float errorRate = __powf(10.0f, -qualityScore / 10.0f);

    // Likelihood calculation considering sequencing error rate
    for (int i = 0; i < 4; i++) { // Consider only ACGT
        if (counts[i] > 0) {
            float expectedErrorRate = __fdividef(errorRate, 3.0f); // Average distribution to other 3 bases
            float observedFreq = __fdividef((float)counts[i], (float)totalDepth);

            // Simplified likelihood calculation
            if (observedFreq > expectedErrorRate) {
                likelihood *= observedFreq;
            } else {
                likelihood *= expectedErrorRate;
            }
        }
    }

    // Bayesian posterior probability = likelihood × prior / evidence
    float posterior = likelihood * priorProb;

    // Normalization (simplified version)
    posteriorProbabilities[gid] = fminf(1.0f, posterior);
}

// CUDA optimized batch processing kernel
__global__ void batchProcessAllelesCUDA(const char* batchPileupData,
                                       const int* batchSizes,
                                       const int* batchOffsets,
                                       int* batchResults,
                                       const int numBatches) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int batchIdx = blockIdx.y;

    if (batchIdx >= numBatches) return;

    int batchSize = batchSizes[batchIdx];
    int offset = batchOffsets[batchIdx];

    if (gid >= batchSize) return;

    // Process each position in the batch
    const char* pileupData = &batchPileupData[offset];
    int* results = &batchResults[batchIdx * batchSize * 6];

    // Execute allele counting
    int counts[6] = {0, 0, 0, 0, 0, 0};

    for (int i = 0; i < batchSize; i++) {
        char base = pileupData[gid * batchSize + i];
        switch(base) {
            case 'A': case 'a': counts[0]++; break;
            case 'C': case 'c': counts[1]++; break;
            case 'G': case 'g': counts[2]++; break;
            case 'T': case 't': counts[3]++; break;
            case '-': counts[4]++; break;
            case '+': counts[5]++; break;
        }
    }

    // Write results
    for (int i = 0; i < 6; i++) {
        results[gid * 6 + i] = counts[i];
    }
}

// CUDA optimized Indel detection and processing kernel
__global__ void processIndelsCUDA(const char* pileupData,
                                 const int* positions,
                                 int* indelInfo,
                                 char* indelSequences,
                                 const int dataLength,
                                 const int numPositions,
                                 const int maxIndelLength) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numPositions) return;

    int startIdx = gid * dataLength;
    int indelCount = 0;
    int indelOutputIdx = gid * maxIndelLength;

    // Scan for indel markers in pileup data
    for (int i = 0; i < dataLength - 1; i++) {
        char current = pileupData[startIdx + i];

        if (current == '+' || current == '-') {
            // Parse indel length
            int indelLen = 0;
            int j = i + 1;

            // Read length digits
            while (j < dataLength && pileupData[startIdx + j] >= '0' && pileupData[startIdx + j] <= '9') {
                indelLen = indelLen * 10 + (pileupData[startIdx + j] - '0');
                j++;
            }

            // Record indel information
            if (indelLen > 0 && indelLen <= maxIndelLength && indelCount < 10) {
                indelInfo[gid * 10 + indelCount * 2] = (current == '+') ? 1 : -1; // Insertion: 1, Deletion: -1
                indelInfo[gid * 10 + indelCount * 2 + 1] = indelLen;

                // Copy indel sequence
                for (int k = 0; k < indelLen && j + k < dataLength; k++) {
                    indelSequences[indelOutputIdx + indelCount * maxIndelLength + k] = pileupData[startIdx + j + k];
                }

                indelCount++;
                i = j + indelLen - 1; // Skip processed sequence
            }
        }
    }

    // Fill remaining positions
    for (int i = indelCount; i < 10; i++) {
        indelInfo[gid * 10 + i * 2] = 0;
        indelInfo[gid * 10 + i * 2 + 1] = 0;
    }
}
